#include "hip/hip_runtime.h"
#include <iostream>
#include "ViewpointEvaluator.h"
#include "TransformFunc.h"
#include "Particle.h"
#include <thrust/device_vector.h>
#include <thrust/count.h>
#include <thrust/execution_policy.h>

texture<float, 3, hipReadModeElementType>  volumeVal;
texture<unsigned short, 3, hipReadModeElementType>  volumeLabel;

texture<float4, 3, hipReadModeElementType>  gradientTexOri;
texture<float4, 3, hipReadModeElementType>  gradientTexFiltered;

ViewpointEvaluator::ViewpointEvaluator(std::shared_ptr<RayCastingParameters> _r, std::shared_ptr<Volume> v)
{
	rcp = std::make_shared<RayCastingParameters>();
	rcp->la = _r->la, rcp->ld = _r->ld, rcp->ls = _r->ls;
	rcp->transFuncP1 = _r->transFuncP1, rcp->transFuncP2 = _r->transFuncP2;
	rcp->density = _r->density;
	rcp->maxSteps = _r->maxSteps;
	rcp->tstep = _r->tstep; rcp->brightness = _r->brightness;
	rcp->useColor = _r->useColor;


	volume = v;

	volumeVal.normalized = false;
	volumeVal.filterMode = hipFilterModeLinear;
	volumeVal.addressMode[0] = hipAddressModeBorder;
	volumeVal.addressMode[1] = hipAddressModeBorder;
	volumeVal.addressMode[2] = hipAddressModeBorder;

	GPU_setConstants(&(rcp->transFuncP1), &(rcp->transFuncP2), &(rcp->la), &(rcp->ld), &(rcp->ls), &(volume->spacing));
	GPU_setVolume(&(volume->volumeCuda));

	rcp->tstep = 1.0; //generally don't need to sample beyond each voxel

	hipMalloc(&d_hist, sizeof(float)*nbins);

	cubeFaceHists.resize(6);
	for (int i = 0; i < 6; i++){
		hipMalloc(&cubeFaceHists[i], sizeof(float)*nbins);
	}

	cubeInfo.resize(6);

	sdkCreateTimer(&timer);

}

void ViewpointEvaluator::createOneParticleFormOfViewSamples()
{
	std::vector<float4> pos;
	std::vector<float> val;
	for (int i = 0; i < skelViews.size(); i++){
		for (int j = 0; j < skelViews[i]->numParticles; j++){
			pos.push_back(skelViews[i]->pos[j]);
			val.push_back(skelViews[i]->val[j]);
		}
	}
	allViewSamples = std::make_shared<Particle>(pos, val);
}

void ViewpointEvaluator::initDownSampledResultVolume(int3 sampleSize)
{
	if (resVol != 0)
		resVol.reset();
	resVol = std::make_shared<Volume>();
	resVol->setSize(sampleSize);

	//note that these two rely on the method to set the viewpoint of the sample. also currently most functions do not consider about the origin
	resVol->dataOrigin = indToLocal(0, 0, 0);
	resVol->spacing = indToLocal(1, 1, 1) - resVol->dataOrigin;
}

float3 ViewpointEvaluator::indToLocal(int i, int j, int k)
{
	return make_float3(i - 1, j - 1, k - 1)*make_float3(volume->size.x, volume->size.y, volume->size.z) / make_float3(resVol->size - 3)*volume->spacing;
}

void ViewpointEvaluator::setLabel(std::shared_ptr<VolumeCUDA> v)
{
	volumeLabel.normalized = false;
	volumeLabel.filterMode = hipFilterModePoint;
	volumeLabel.addressMode[0] = hipAddressModeBorder;
	volumeLabel.addressMode[1] = hipAddressModeBorder;
	volumeLabel.addressMode[2] = hipAddressModeBorder;

	checkCudaErrors(hipBindTextureToArray(volumeLabel, v->content, v->channelDesc));

	labelBeenSet = true;
}

void ViewpointEvaluator::initLabelVisibility()
{
	if (LabelVisibilityInited)	return;

	if (!labelBeenSet){
		std::cout << "label volume not set for the viewpoint evaluator! " << std::endl;
		exit(0);
	}

	if (d_r != 0) hipFree(d_r);
	setSpherePoints();
	hipMalloc(&d_r, sizeof(float)*numSphereSample);
	JS06SphereInited = false;
	Tao09DetailInited = false;
	LabelVisibilityInited = true;
}

void ViewpointEvaluator::initJS06Sphere()
{
	if (JS06SphereInited)	return;

	if (d_r != 0) hipFree(d_r);
	setSpherePoints();
	hipMalloc(&d_r, sizeof(float)*numSphereSample);
	JS06SphereInited = true;
	Tao09DetailInited = false;
	LabelVisibilityInited = false;
}

void ViewpointEvaluator::initTao09Detail()
{
	if (Tao09DetailInited)	return;

	std::cout << "initializing for viewpoint evaluation method" << std::endl;

	if (d_r != 0) hipFree(d_r);
	setSpherePoints();
	hipMalloc(&d_r, sizeof(float)*numSphereSample);
	
	if (!noBilat){
		float* gradient = 0;
		volume->computeGradient(gradient);
		volumeGradient.VolumeCUDA_deinit();
		volumeGradient.VolumeCUDA_init(volume->size, gradient, 0, 4);
		delete[] gradient;

		float* bilateralVolumeRes = new float[volume->size.x*volume->size.y*volume->size.z];
		FILE * fp = fopen((dataFolder + "/bilat.raw").c_str(), "rb");
		fread(bilateralVolumeRes, sizeof(float), volume->size.x*volume->size.y*volume->size.z, fp);
		fclose(fp);

		float* bGradient = 0;
		volume->computeGradient(bilateralVolumeRes, volume->size, bGradient);
		filteredVolumeGradient.VolumeCUDA_deinit();
		filteredVolumeGradient.VolumeCUDA_init(volume->size, bGradient, 0, 4);
		delete[] bGradient;
		delete[] bilateralVolumeRes;
	}

	gradientTexOri.normalized = false;
	gradientTexOri.filterMode = hipFilterModeLinear;
	gradientTexOri.addressMode[0] = hipAddressModeBorder;
	gradientTexOri.addressMode[1] = hipAddressModeBorder;
	gradientTexOri.addressMode[2] = hipAddressModeBorder;
	
	gradientTexFiltered.normalized = false;
	gradientTexFiltered.filterMode = hipFilterModeLinear;
	gradientTexFiltered.addressMode[0] = hipAddressModeBorder;
	gradientTexFiltered.addressMode[1] = hipAddressModeBorder;
	gradientTexFiltered.addressMode[2] = hipAddressModeBorder;
	
	Tao09DetailInited = true;
	JS06SphereInited = false;
	LabelVisibilityInited = false;
}

void ViewpointEvaluator::compute_UniformSampling(VPMethod m)
{
	maxEntropy = -999;
	int3 sampleSize = resVol->size;
	if (m == BS05){
	}
	else if (m == JS06Sphere){
		//initJS06Sphere();
		//for (int k = 0; k < sampleSize.z; k++){
		//	std::cout << "now doing k = " << k << std::endl;
		//	for (int j = 0; j < sampleSize.y; j++){
		//		for (int i = 0; i < sampleSize.x; i++){
		//			float3 eyeInLocal = indToLocal(i, j, k);
		//			float entroRes = computeEntropyJS06Sphere(eyeInLocal);
		//			resVol->values[k*sampleSize.y*sampleSize.x + j*sampleSize.x + i] = entroRes;
		//			if (entroRes>maxEntropy){
		//				maxEntropy = entroRes;
		//				optimalEyeInLocal = eyeInLocal;
		//			}
		//		}
		//	}
		//}
	}
}

void ViewpointEvaluator::compute_NextSkelSampling(VPMethod m) // !!!NOTE !!! currently only work for m==Tao09
{
	if (lastSkelOfOptimal<0 || lastSkelOfOptimal>=skelViews.size())
		return;

	skelViewsConsidered[lastSkelOfOptimal] = false;
	compute_SkelSampling(m);
}

void ViewpointEvaluator::compute_SkelSampling(VPMethod m)
{
	if ((m == JS06Sphere && !JS06SphereInited) || (m == LabelVisibility && !LabelVisibilityInited) || (
		m == Tao09Detail && !Tao09DetailInited)){
		std::cout << "NOTE! time cost for computing the global optimal includes initiation time: " << std::endl;
	}


	sdkResetTimer(&timer);
	sdkStartTimer(&timer);

	maxEntropy = -999;
	//int3 sampleSize = resVol->size;
	if (m == JS06Sphere){
		initJS06Sphere();
		for (int i = 0; i < skelViews.size(); i++){
			for (int j = 0; j < skelViews[i]->numParticles; j++){
				float3 eyeInLocal = make_float3(skelViews[i]->pos[j]);
				float entroRes = computeLocalSphereEntropy(eyeInLocal, JS06Sphere);
				if (entroRes>maxEntropy){
					maxEntropy = entroRes;
					optimalEyeInLocal = eyeInLocal;
					lastSkelOfOptimal = i;
				}
			}
		}
	}
	else if (m == LabelVisibility){
		initLabelVisibility();
		for (int i = 0; i < skelViews.size(); i++){
			for (int j = 0; j < skelViews[i]->numParticles; j++){
				float3 eyeInLocal = make_float3(skelViews[i]->pos[j]);
				float entroRes = computeLocalSphereEntropy(eyeInLocal, LabelVisibility);
				if (entroRes>maxEntropy){
					maxEntropy = entroRes;
					optimalEyeInLocal = eyeInLocal;
					lastSkelOfOptimal = i;
				}
			}
		}
	}
	else if (m == Tao09Detail){
		if (noBilat){
			return;
		}

		initTao09Detail();

		checkCudaErrors(hipBindTextureToArray(gradientTexOri, volumeGradient.content, volumeGradient.channelDesc));
		checkCudaErrors(hipBindTextureToArray(gradientTexFiltered, filteredVolumeGradient.content, filteredVolumeGradient.channelDesc));

		for (int i = 0; i < skelViews.size(); i++){
			if (!skelViewsConsidered[i])
				continue; 
			for (int j = 0; j < skelViews[i]->numParticles; j++){
				float3 eyeInLocal = make_float3(skelViews[i]->pos[j]);
				float entroRes = computeLocalSphereEntropy(eyeInLocal, Tao09Detail);
				if (entroRes>maxEntropy){
					maxEntropy = entroRes;
					optimalEyeInLocal = eyeInLocal;
					lastSkelOfOptimal = i;
				}
			}
		}
		
		checkCudaErrors(hipUnbindTexture(gradientTexOri));
		checkCudaErrors(hipUnbindTexture(gradientTexFiltered));
	}

	sdkStopTimer(&timer);

	float timeCost = sdkGetAverageTimerValue(&timer) / 1000.f;
	std::cout << "time cost for computing the global optimal: " << timeCost <<std::endl;

}

void ViewpointEvaluator::saveResultVol(const char* fname)
{
	resVol->saveRawToFile(fname);
}

void ViewpointEvaluator::setSpherePoints(int n)
{
	if (spherePointSet) return;

	//source: https://www.openprocessing.org/sketch/41142

	numSphereSample = n;
	sphereSamples.resize(n);

	float phi = (sqrt(5) + 1) / 2 - 1; // golden ratio
	float ga = phi * 2 * M_PI;           // golden angle

	for (int i = 1; i <= numSphereSample; ++i) {
		float lon = ga*i;
		lon /= 2 * M_PI; lon -= floor(lon); lon *= 2 * M_PI;
		if (lon > M_PI)  lon -= 2 * M_PI;

		// Convert dome height (which is proportional to surface area) to latitude
		float lat = asin(-1 + 2 * i / (float)numSphereSample);

		sphereSamples[i - 1] = SpherePoint(lat, lon);
	}
	if (d_sphereSamples != 0){
		hipFree(d_sphereSamples);
	}
	hipMalloc(&d_sphereSamples, sizeof(float)*numSphereSample * 3);
	hipMemcpy(d_sphereSamples, (float*)(&sphereSamples[0]), sizeof(float)*numSphereSample * 3, hipMemcpyHostToDevice);
	spherePointSet = true;
}

__constant__ float colorTableDiverge[33][4] = {
	0, 0.231372549, 0.298039216, 0.752941176,
	0.03125, 0.266666667, 0.352941176, 0.8,
	0.0625, 0.301960784, 0.407843137, 0.843137255,
	0.09375, 0.341176471, 0.458823529, 0.882352941,
	0.125, 0.384313725, 0.509803922, 0.917647059,
	0.15625, 0.423529412, 0.556862745, 0.945098039,
	0.1875, 0.466666667, 0.603921569, 0.968627451,
	0.21875, 0.509803922, 0.647058824, 0.984313725,
	0.25, 0.552941176, 0.690196078, 0.996078431,
	0.28125, 0.596078431, 0.725490196, 1,
	0.3125, 0.639215686, 0.760784314, 1,
	0.34375, 0.682352941, 0.788235294, 0.992156863,
	0.375, 0.721568627, 0.815686275, 0.976470588,
	0.40625, 0.760784314, 0.835294118, 0.956862745,
	0.4375, 0.8, 0.850980392, 0.933333333,
	0.46875, 0.835294118, 0.858823529, 0.901960784,
	0.5, 0.866666667, 0.866666667, 0.866666667,
	0.53125, 0.898039216, 0.847058824, 0.819607843,
	0.5625, 0.925490196, 0.82745098, 0.77254902,
	0.59375, 0.945098039, 0.8, 0.725490196,
	0.625, 0.960784314, 0.768627451, 0.678431373,
	0.65625, 0.968627451, 0.733333333, 0.62745098,
	0.6875, 0.968627451, 0.694117647, 0.580392157,
	0.71875, 0.968627451, 0.650980392, 0.529411765,
	0.75, 0.956862745, 0.603921569, 0.482352941,
	0.78125, 0.945098039, 0.552941176, 0.435294118,
	0.8125, 0.925490196, 0.498039216, 0.388235294,
	0.84375, 0.898039216, 0.439215686, 0.345098039,
	0.875, 0.870588235, 0.376470588, 0.301960784,
	0.90625, 0.835294118, 0.31372549, 0.258823529,
	0.9375, 0.796078431, 0.243137255, 0.219607843,
	0.96875, 0.752941176, 0.156862745, 0.184313725,
	1, 0.705882353, 0.015686275, 0.149019608,
};

__device__ float3 GetColourDiverge2(float v)
{
	//can be accelerated using binary search!!
	int pos = 0;
	bool notFound = true;
	const int numItemColorTableDiverge = 33;
	while (pos < numItemColorTableDiverge - 1 && notFound) {
		if (colorTableDiverge[pos][0] <= v && colorTableDiverge[pos + 1][0] >= v)
			notFound = false;
		pos++;
	}
	float ratio = (v - colorTableDiverge[pos][0]) / (colorTableDiverge[pos + 1][0] - colorTableDiverge[pos][0]);
	
	float3 c = make_float3(
		ratio*(colorTableDiverge[pos + 1][1] - colorTableDiverge[pos][1]) + colorTableDiverge[pos][1],
		ratio*(colorTableDiverge[pos + 1][2] - colorTableDiverge[pos][2]) + colorTableDiverge[pos][2],
		ratio*(colorTableDiverge[pos + 1][3] - colorTableDiverge[pos][3]) + colorTableDiverge[pos][3]);

	return(c);
}


__device__
int intersectBox2(Ray r, float3 boxmin, float3 boxmax, float *tnear, float *tfar)
{
	// compute intersection of ray with all six bbox planes
	float3 invR = make_float3(1.0f) / r.d;
	float3 tbot = invR * (boxmin - r.o);
	float3 ttop = invR * (boxmax - r.o);

	// re-order intersections to find smallest and largest on each axis
	float3 tmin = fminf(ttop, tbot);
	float3 tmax = fmaxf(ttop, tbot);

	// find the largest tmin and the smallest tmax
	float largest_tmin = fmaxf(fmaxf(tmin.x, tmin.y), fmaxf(tmin.x, tmin.z));
	float smallest_tmax = fminf(fminf(tmax.x, tmax.y), fminf(tmax.x, tmax.z));

	*tnear = largest_tmin;
	*tfar = smallest_tmax;

	return smallest_tmax > largest_tmin;
}



__constant__ float transFuncP1;
__constant__ float transFuncP2;
__constant__ float la;
__constant__ float ld;
__constant__ float ls;
__constant__ float3 spacing;

void ViewpointEvaluator::GPU_setVolume(const VolumeCUDA *vol)
{
	checkCudaErrors(hipBindTextureToArray(volumeVal, vol->content, vol->channelDesc));
}


void ViewpointEvaluator::GPU_setConstants(float* _transFuncP1, float* _transFuncP2, float* _la, float* _ld, float* _ls, float3* _spacing)
{
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(transFuncP1), _transFuncP1, sizeof(float)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(transFuncP2), _transFuncP2, sizeof(float)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(la), _la, sizeof(float)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(ld), _ld, sizeof(float)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(ls), _ls, sizeof(float)));

	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(spacing), _spacing, sizeof(float3)));
}


struct functor_computeEntropy
{
	float sum;
	__device__ __host__ float operator() (float r)
	{
		if (r < 0.00001){
			return 0;
		}
		else{
			float qj = r / sum;
			return -qj*log(qj);
		}
	}
	functor_computeEntropy(float s) : sum(s){}
};





__global__ void d_computeSphereColor(float density, float brightness,
	float3 eyeInLocal, int3 volumeSize, int maxSteps, float tstep, bool useColor, float * r, int numSphereSample, float *sphereSamples, float *hist, int nbins, bool useHist, VPMethod vpmethod)
{

	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i >= numSphereSample)	return;

	const float opacityThreshold = 0.95f;


	Ray eyeRay;
	eyeRay.o = eyeInLocal;
	eyeRay.d = make_float3(sphereSamples[3 * i], sphereSamples[3 * i + 1], sphereSamples[3 * i + 2]);

	float tnear, tfar;
	const float3 boxMin = make_float3(0.0f, 0.0f, 0.0f);
	const float3 boxMax = spacing*make_float3(volumeSize); 
	intersectBox2(eyeRay, boxMin, boxMax, &tnear, &tfar);
	tnear = 0.01f;	//!!!NOTE!!! this tnear is not in the clip space but in the original space

	// march along ray from front to back, accumulating color
	float t = tnear;
	float3 pos = eyeRay.o + eyeRay.d*tnear;
	float3 step = eyeRay.d*tstep;

	float4 sum = make_float4(0.0f); //for JS
	unsigned short label = 0; //for label count
	float detailDescriptor = 0; //for TaoDetail

	float lightingThr = 0.000001; //used for the threshold of TaoDetail

	for (int i = 0; i<maxSteps; i++)
	{
		float3 coord = pos / spacing;
		float sample = tex3D(volumeVal, coord.x, coord.y, coord.z);
		float funcRes = clamp((sample - transFuncP2) / (transFuncP1 - transFuncP2), 0.0, 1.0);

		// lookup in transfer function texture
		float4 col;

		float3 cc;
		if (useColor)
			cc = GetColourDiverge2(clamp(funcRes, 0.0f, 1.0f));
		else
			cc = make_float3(funcRes, funcRes, funcRes);

		////currently ignore lighting
		col = make_float4(la*cc, funcRes);

		col.w *= density;

		// pre-multiply alpha
		col.x *= col.w;
		col.y *= col.w;
		col.z *= col.w;
		
		float visibility = 1.0f - sum.w;
		
		// "over" operator for front-to-back blending
		sum = sum + col*(1.0f - sum.w);

		if (vpmethod == Tao09Detail){ //if not Tao09Detail, the sampled texture may not be prepared
			float curDetail = 0;
			float3 normalOri = make_float3(tex3D(gradientTexOri, coord.x, coord.y, coord.z)) / spacing;
			float3 normalFiltered = make_float3(tex3D(gradientTexFiltered, coord.x, coord.y, coord.z)) / spacing;
			if (length(normalOri) > lightingThr){
				if (length(normalFiltered) > lightingThr){
					curDetail = 1 - dot(normalize(normalOri), normalize(normalFiltered));
				}
				else{
					curDetail = 1;
				}
			}
			detailDescriptor = detailDescriptor + curDetail*col.w*visibility;
		}

		// exit early if opaque
		if (sum.w > opacityThreshold){
			break;
		}
		else if (vpmethod == LabelVisibility){
			unsigned short curlabel = tex3D(volumeLabel, coord.x, coord.y, coord.z);
			if (curlabel > label)
			{
				label = curlabel;
			}
		}

		t += tstep;

		if (t > tfar){
			break;
		}

		pos += step;
	}

	sum *= brightness;

	float uv = sum.x;
	r[i] = uv;

	if (vpmethod == Tao09Detail)
	{
		uv = detailDescriptor;
		r[i] = uv;
		// !!! this is true only when we know uv is in [0,2] !!!
		int bin = min((int)((uv/2)*nbins), nbins - 1);
		atomicAdd(hist + bin, 1);
	}
	else if (vpmethod == LabelVisibility){
		r[i] = label;
		if (useHist){
			int bin;
			if (label > 0)
				bin = 1;
			else
				bin = 0;
			atomicAdd(hist + bin, 1);
		}
	}
	else{
		if (useHist){
			// !!! this is true only when we know uv is in [0,1] !!!
			int bin = min((int)(uv*nbins), nbins - 1);
			atomicAdd(hist + bin, 1);
		}
	}
}

//for certain method color is not needed. only use density to control when to stop the integration
__global__ void d_computeSphereNoColor(float density,
	float3 eyeInLocal, int3 volumeSize, int maxSteps, float tstep, float * r, int numSphereSample, float *sphereSamples, float *hist, int nbins, bool useHist, VPMethod vpmethod)
{

	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i >= numSphereSample)	return;

	const float opacityThreshold = 0.95f;


	Ray eyeRay;
	eyeRay.o = eyeInLocal;
	eyeRay.d = make_float3(sphereSamples[3 * i], sphereSamples[3 * i + 1], sphereSamples[3 * i + 2]);

	float tnear, tfar;
	const float3 boxMin = make_float3(0.0f, 0.0f, 0.0f);
	const float3 boxMax = spacing*make_float3(volumeSize);
	intersectBox2(eyeRay, boxMin, boxMax, &tnear, &tfar);
	tnear = 0.01f;	//!!!NOTE!!! this tnear is not in the clip space but in the original space

	// march along ray from front to back, accumulating color
	float t = tnear;
	float3 pos = eyeRay.o + eyeRay.d*tnear;
	float3 step = eyeRay.d*tstep;

	float4 sum = make_float4(0.0f); //for JS
	unsigned short label = 0; //for label count
	float detailDescriptor = 0; //for TaoDetail

	float lightingThr = 0.000001; //used for the threshold of TaoDetail

	for (int i = 0; i<maxSteps; i++)
	{
		float3 coord = pos / spacing;
		float sample = tex3D(volumeVal, coord.x, coord.y, coord.z);
		float funcRes = clamp((sample - transFuncP2) / (transFuncP1 - transFuncP2), 0.0, 1.0);

		float colDensity = funcRes;
		float4 col = make_float4(0, 0, 0, colDensity); //(0,0,0) as fake color

		col.w *= density;

		float visibility = 1.0f - sum.w;

		// "over" operator for front-to-back blending
		sum = sum + col*(1.0f - sum.w);

		if (vpmethod == Tao09Detail){ //if not Tao09Detail, the sampled texture may not be prepared
			float curDetail = 0;
			float3 normalOri = make_float3(tex3D(gradientTexOri, coord.x, coord.y, coord.z)) / spacing;
			float3 normalFiltered = make_float3(tex3D(gradientTexFiltered, coord.x, coord.y, coord.z)) / spacing;
			if (length(normalOri) > lightingThr){
				if (length(normalFiltered) > lightingThr){
					curDetail = 1 - dot(normalize(normalOri), normalize(normalFiltered));
				}
				else{
					curDetail = 1;
				}
			}
			detailDescriptor = detailDescriptor + curDetail*col.w*visibility;
		}

		// exit early if opaque
		if (sum.w > opacityThreshold){
			break;
		}
		else if (vpmethod == LabelVisibility){
			unsigned short curlabel = tex3D(volumeLabel, coord.x, coord.y, coord.z);
			if (curlabel > label)
			{
				label = curlabel;
			}
		}

		t += tstep;

		if (t > tfar){
			break;
		}

		pos += step;
	}

	if (vpmethod == Tao09Detail)
	{
		float uv = detailDescriptor;
		r[i] = uv;
		// !!! this is true only when we know uv is in [0,2] !!!
		int bin = min((int)((uv / 2)*nbins), nbins - 1);
		atomicAdd(hist + bin, 1);
	}
	else if (vpmethod == LabelVisibility){
		r[i] = label;
		if (useHist){
			int bin;
			bin = label;
			//if (label > 0)
			//	bin = 1;
			//else
			//	bin = 0;
			atomicAdd(hist + bin, 1);
		}
	}
	else{
//should be error
		r[i] = -789; //for debug
	}
}

struct is_solid
{
	__host__ __device__
	bool operator()(float x)
	{
		return x>0.00001;
	}
};


float ViewpointEvaluator::computeLocalSphereEntropy(float3 eyeInLocal, VPMethod m)
{
	int threadsPerBlock = 64;
	int blocksPerGrid = (numSphereSample + threadsPerBlock - 1) / threadsPerBlock;

	hipMemset(d_hist, 0, sizeof(float)*nbins);

	//d_computeSphereColor << <blocksPerGrid, threadsPerBlock >> >(rcp->density, rcp->brightness, eyeInLocal, volume->size, rcp->maxSteps, rcp->tstep, rcp->useColor, d_r, numSphereSample, d_sphereSamples, d_hist, nbins, useHist, m);
	d_computeSphereNoColor << <blocksPerGrid, threadsPerBlock >> >(rcp->density, eyeInLocal, volume->size, rcp->maxSteps, rcp->tstep, d_r, numSphereSample, d_sphereSamples, d_hist, nbins, useHist, m);

	float ret;
	if (useHist){
		if (m == LabelVisibility){
			ret = computeVectorEntropy(d_hist, maxLabel + 1);
		}
		else {
			ret = computeVectorEntropy(d_hist, nbins);
		}
	}
	else{//not finished yet
		ret = computeVectorEntropy(d_r, numSphereSample);
	}

	return ret;
}

float ViewpointEvaluator::computeVectorEntropy(float* ary, int size)
{
	thrust::device_vector< float > iVec(ary, ary + size);

	//for debug
	std::vector<float> stl_vector(size);
	thrust::copy(iVec.begin(), iVec.end(), stl_vector.begin());

	float sum = thrust::reduce(iVec.begin(), iVec.end(), (float)0, thrust::plus<float>());
	thrust::transform(iVec.begin(), iVec.end(), iVec.begin(), functor_computeEntropy(sum));
	return thrust::reduce(iVec.begin(), iVec.end(), (float)0, thrust::plus<float>());
}


__global__ void d_computeCubeColorHist(float density, float brightness,
	float3 eyeInLocal, float3 viewVec, float3 upVec, int3 volumeSize, int maxSteps, float tstep, bool useColor, float * r, int numSphereSample, float *sphereSamples, float *hist0, float *hist1, float *hist2, float *hist3, float *hist4, float *hist5, int nbins, bool useHist, VPMethod vpmethod)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i >= numSphereSample)	return;

	const float opacityThreshold = 0.95f;

	Ray eyeRay;
	eyeRay.o = eyeInLocal;
	eyeRay.d = make_float3(sphereSamples[3 * i], sphereSamples[3 * i + 1], sphereSamples[3 * i + 2]);

	float tnear, tfar;
	const float3 boxMin = make_float3(0.0f, 0.0f, 0.0f);
	const float3 boxMax = spacing*make_float3(volumeSize);
	intersectBox2(eyeRay, boxMin, boxMax, &tnear, &tfar);
	tnear = 0.01f;	//!!!NOTE!!! this tnear is not in the clip space but in the original space

	// march along ray from front to back, accumulating color
	float t = tnear;
	float3 pos = eyeRay.o + eyeRay.d*tnear;
	float3 step = eyeRay.d*tstep;

	float4 sum = make_float4(0.0f); //for JS
	unsigned short label = 0; //for label count
	float detailDescriptor= 0; //for TaoDetail

	float lightingThr = 0.000001; //used for the threshold of TaoDetail


	for (int i = 0; i<maxSteps; i++)
	{
		float3 coord = pos / spacing;
		float sample = tex3D(volumeVal, coord.x, coord.y, coord.z);
		float funcRes = clamp((sample - transFuncP2) / (transFuncP1 - transFuncP2), 0.0, 1.0);

		// lookup in transfer function texture
		float4 col;

		float3 cc;
		if (useColor)
			cc = GetColourDiverge2(clamp(funcRes, 0.0f, 1.0f));
		else
			cc = make_float3(funcRes, funcRes, funcRes);

		////currently ignore lighting
		col = make_float4(la*cc, funcRes);

		col.w *= density;

		// pre-multiply alpha
		col.x *= col.w;
		col.y *= col.w;
		col.z *= col.w;
		// "over" operator for front-to-back blending

		float visibility = 1.0f - sum.w;

		sum = sum + col*(1.0f - sum.w);


		if (vpmethod == Tao09Detail){ //if not Tao09Detail, the sampled texture may not be prepared
			float curDetail = 0;
			float3 normalOri = make_float3(tex3D(gradientTexOri, coord.x, coord.y, coord.z)) / spacing;
			float3 normalFiltered = make_float3(tex3D(gradientTexFiltered, coord.x, coord.y, coord.z)) / spacing;
			if (length(normalOri) > lightingThr){
				if (length(normalFiltered) > lightingThr){
					curDetail = 1 - dot(normalize(normalOri), normalize(normalFiltered));
				}
				else{
					curDetail = 1;
				}
			}
			detailDescriptor = detailDescriptor + curDetail*col.w*visibility;
		}

		// exit early if opaque
		if (sum.w > opacityThreshold){
			break;
		}
		else if (vpmethod == LabelVisibility){
			unsigned short curlabel = tex3D(volumeLabel, coord.x, coord.y, coord.z);
			if (curlabel > label)
			{
				label = curlabel;
			}
		}

		t += tstep;
		if (t > tfar){
			break;
		}
		pos += step;
	}
	sum *= brightness;

	int bin;	
	float uv = sum.x;
	r[i] = uv;
	if (vpmethod == Tao09Detail)
	{
		uv = detailDescriptor;
		r[i] = uv;
		// !!! this is true only when we know uv is in [0,2] !!!
		bin = min((int)((uv / 2)*nbins), nbins - 1);
	}
	else if (vpmethod == LabelVisibility){
		r[i] = label;
		if (useHist){
			if (label > 0)
				bin = 1;
			else
				bin = 0;
		}
	}
	else{
		if (useHist){
			// !!! this is true only when we know uv is in [0,1] !!!
			bin = min((int)(uv*nbins), nbins - 1);
		}
	}

	//suppose x coord is along viewVew, suppose upVec and viewVew are normalized and perpendicular
	float3 sidevec = cross(upVec, viewVec);
	float rayz = dot(eyeRay.d, upVec), rayx = dot(eyeRay.d, viewVec), rayy = dot(eyeRay.d, sidevec);

	float xabs = abs(rayx), yabs = abs(rayy), zabs = abs(rayz);
	if (xabs > yabs && xabs > zabs){
		if (rayx > 0){ //front
			atomicAdd(hist0 + bin, 1);
		}
		else{ //back
			atomicAdd(hist1 + bin, 1);
		}
	}
	else if (yabs > xabs && yabs > zabs){
		if (rayy > 0){ //left
			atomicAdd(hist2 + bin, 1);
		}
		else{ //right
			atomicAdd(hist3 + bin, 1);
		}
	}
	else{ //zabs is the max
		if (rayz > 0){ //up
			atomicAdd(hist4 + bin, 1);
		}
		else{ //below
			atomicAdd(hist5 + bin, 1);
		}
	}
}


__global__ void d_computeCubeNoColorHist(float density, float3 eyeInLocal, float3 viewVec, float3 upVec, int3 volumeSize, int maxSteps, float tstep, float * r, int numSphereSample, float *sphereSamples, float *hist0, float *hist1, float *hist2, float *hist3, float *hist4, float *hist5, int nbins, bool useHist, VPMethod vpmethod)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i >= numSphereSample)	return;

	const float opacityThreshold = 0.95f;

	Ray eyeRay;
	eyeRay.o = eyeInLocal;
	eyeRay.d = make_float3(sphereSamples[3 * i], sphereSamples[3 * i + 1], sphereSamples[3 * i + 2]);

	float tnear, tfar;
	const float3 boxMin = make_float3(0.0f, 0.0f, 0.0f);
	const float3 boxMax = spacing*make_float3(volumeSize);
	intersectBox2(eyeRay, boxMin, boxMax, &tnear, &tfar);
	tnear = 0.01f;	//!!!NOTE!!! this tnear is not in the clip space but in the original space

	// march along ray from front to back, accumulating color
	float t = tnear;
	float3 pos = eyeRay.o + eyeRay.d*tnear;
	float3 step = eyeRay.d*tstep;

	float4 sum = make_float4(0.0f); //for JS
	unsigned short label = 0; //for label count
	float detailDescriptor = 0; //for TaoDetail

	float lightingThr = 0.000001; //used for the threshold of TaoDetail


	for (int i = 0; i<maxSteps; i++)
	{
		float3 coord = pos / spacing;
		float sample = tex3D(volumeVal, coord.x, coord.y, coord.z);
		float funcRes = clamp((sample - transFuncP2) / (transFuncP1 - transFuncP2), 0.0, 1.0);

		float colDensity = funcRes;
		float4 col = make_float4(0,0,0, colDensity);

		col.w *= density;

		float visibility = 1.0f - sum.w;

		sum = sum + col*(1.0f - sum.w);

		if (vpmethod == Tao09Detail){ //if not Tao09Detail, the sampled texture may not be prepared
			float curDetail = 0;
			float3 normalOri = make_float3(tex3D(gradientTexOri, coord.x, coord.y, coord.z)) / spacing;
			float3 normalFiltered = make_float3(tex3D(gradientTexFiltered, coord.x, coord.y, coord.z)) / spacing;
			if (length(normalOri) > lightingThr){
				if (length(normalFiltered) > lightingThr){
					curDetail = 1 - dot(normalize(normalOri), normalize(normalFiltered));
				}
				else{
					curDetail = 1;
				}
			}
			detailDescriptor = detailDescriptor + curDetail*col.w*visibility;
		}

		// exit early if opaque
		if (sum.w > opacityThreshold){
			break;
		}
		else if (vpmethod == LabelVisibility){
			unsigned short curlabel = tex3D(volumeLabel, coord.x, coord.y, coord.z);
			if (curlabel > label)
			{
				label = curlabel;
			}
		}

		t += tstep;
		if (t > tfar){
			break;
		}
		pos += step;
	}

	int bin;

	if (vpmethod == Tao09Detail)
	{
		float uv = detailDescriptor;
		r[i] = uv;
		// !!! this is true only when we know uv is in [0,2] !!!
		bin = min((int)((uv / 2)*nbins), nbins - 1);
	}
	else if (vpmethod == LabelVisibility){
		r[i] = label;
		if (useHist){
			//if (label > 0)
			//	bin = 1;
			//else
			//	bin = 0;
			bin = label;
		}
	}
	else{
		//should be error
	}

	//suppose x coord is along viewVew, suppose upVec and viewVew are normalized and perpendicular
	float3 sidevec = cross(upVec, viewVec);
	float rayz = dot(eyeRay.d, upVec), rayx = dot(eyeRay.d, viewVec), rayy = dot(eyeRay.d, sidevec);

	float xabs = abs(rayx), yabs = abs(rayy), zabs = abs(rayz);
	if (xabs > yabs && xabs > zabs){
		if (rayx > 0){ //front
			atomicAdd(hist0 + bin, 1);
		}
		else{ //back
			atomicAdd(hist1 + bin, 1);
		}
	}
	else if (yabs > xabs && yabs > zabs){
		if (rayy > 0){ //left
			atomicAdd(hist2 + bin, 1);
		}
		else{ //right
			atomicAdd(hist3 + bin, 1);
		}
	}
	else{ //zabs is the max
		if (rayz > 0){ //up
			atomicAdd(hist4 + bin, 1);
		}
		else{ //below
			atomicAdd(hist5 + bin, 1);
		}
	}
}

void ViewpointEvaluator::computeCubeEntropy(float3 eyeInLocal, float3 viewDir, float3 upDir, VPMethod m)
{
	if (m == Tao09Detail){
		if (noBilat){
			return;
		}

		initTao09Detail();

		int threadsPerBlock = 64;
		int blocksPerGrid = (numSphereSample + threadsPerBlock - 1) / threadsPerBlock;

		for (int i = 0; i < 6; i++){
			hipMemset(cubeFaceHists[i], 0, sizeof(float)*nbins);
		}

		checkCudaErrors(hipBindTextureToArray(gradientTexOri, volumeGradient.content, volumeGradient.channelDesc));
		checkCudaErrors(hipBindTextureToArray(gradientTexFiltered, filteredVolumeGradient.content, filteredVolumeGradient.channelDesc));

	//	d_computeCubeColorHist << <blocksPerGrid, threadsPerBlock >> >(rcp->density, rcp->brightness, eyeInLocal, viewDir, upDir, volume->size, rcp->maxSteps, rcp->tstep, rcp->useColor, d_r, numSphereSample, d_sphereSamples, cubeFaceHists[0], cubeFaceHists[1], cubeFaceHists[2], cubeFaceHists[3], cubeFaceHists[4], cubeFaceHists[5], nbins, useHist, m);
		d_computeCubeNoColorHist << <blocksPerGrid, threadsPerBlock >> >(rcp->density, eyeInLocal, viewDir, upDir, volume->size, rcp->maxSteps, rcp->tstep, d_r, numSphereSample, d_sphereSamples, cubeFaceHists[0], cubeFaceHists[1], cubeFaceHists[2], cubeFaceHists[3], cubeFaceHists[4], cubeFaceHists[5], nbins, useHist, m);

		checkCudaErrors(hipUnbindTexture(gradientTexOri));
		checkCudaErrors(hipUnbindTexture(gradientTexFiltered));

		for (int i = 0; i < 6; i++){
			if (useHist){
				if (m == LabelVisibility){
					cubeInfo[i] = computeVectorEntropy(cubeFaceHists[i], maxLabel + 1);
				}
				else{
					cubeInfo[i] = computeVectorEntropy(cubeFaceHists[i], nbins);
				}
			}
			else{
				std::cout << "entropy computation not defined! " << std::endl;
				exit(0);
			}
		}
	}
	else if (m == LabelVisibility){
		initLabelVisibility();

		int threadsPerBlock = 64;
		int blocksPerGrid = (numSphereSample + threadsPerBlock - 1) / threadsPerBlock;

		for (int i = 0; i < 6; i++){
			hipMemset(cubeFaceHists[i], 0, sizeof(float)*nbins);
		}

		//	d_computeCubeColorHist << <blocksPerGrid, threadsPerBlock >> >(rcp->density, rcp->brightness, eyeInLocal, viewDir, upDir, volume->size, rcp->maxSteps, rcp->tstep, rcp->useColor, d_r, numSphereSample, d_sphereSamples, cubeFaceHists[0], cubeFaceHists[1], cubeFaceHists[2], cubeFaceHists[3], cubeFaceHists[4], cubeFaceHists[5], nbins, useHist, m);
		d_computeCubeNoColorHist << <blocksPerGrid, threadsPerBlock >> >(rcp->density, eyeInLocal, viewDir, upDir, volume->size, rcp->maxSteps, rcp->tstep, d_r, numSphereSample, d_sphereSamples, cubeFaceHists[0], cubeFaceHists[1], cubeFaceHists[2], cubeFaceHists[3], cubeFaceHists[4], cubeFaceHists[5], nbins, useHist, m);

		for (int i = 0; i < 6; i++){
			if (useHist){
				if (m == LabelVisibility){
					cubeInfo[i] = computeVectorEntropy(cubeFaceHists[i], maxLabel + 1);
				}
				else{
					cubeInfo[i] = computeVectorEntropy(cubeFaceHists[i], nbins);
				}
			}
			else{
				std::cout << "entropy computation not defined! " << std::endl;
				exit(0);
			}
		}
	}
	else{
		return;
	}

}

